#include "AraxLibUtilsGPU.h"
#include "arax.h"
#include "core/arax_data.h"
#include "core/arax_data_private.h"
#include "cuda_utils.h"
#include <atomic>
#include <condition_variable>
#include <sstream>

#define RED "\033[1;31m"
#define GREEN "\033[1;32m"
#define RESET "\033[0m"

bool print_once = true;

/* Enable SYNC or ASYNC in GPU transfers*/
//#define SYNC_H2D_TRANSFERS

//#define BREAKDOWNS_CONTROLLER
// define SYNC to take meassurments
#ifdef BREAKDOWNS_CONTROLLER
#define SYNC() deviceSynchronize()

void deviceSynchronize() {
  if (print_once) {
    cerr << " ========================================================="
         << std::endl;
    cerr << RED << "Async calls are not supported due to meassurments!!"
         << RESET << endl;
    cerr << "For performance disable breakdowns_controller in ccmake!" << endl;
    cerr << " ========================================================="
         << std::endl;
    print_once = false;
  }
  hipDeviceSynchronize();
}
#else
#define SYNC()
#endif

/* Enable the same flag in caffe (src/caffe/syncedmem.cpp)
   to check that allocations match with free */
//#define DATA_LEAKS
#ifdef DATA_LEAKS
#define PRINT_DATA_LEAKS(data) printLeaks(data, __func__, __FILE__, __LINE__)
void printLeaks(arax_data_s *data, const char *func, const char *file,
                size_t line) {
  cerr << "Func: " << func << " ,Size: " << arax_data_size(data)
       << " ,vdata: " << data << " ,remote data " << data->remote << endl;
}
#else
#define PRINT_DATA_LEAKS(data)
#endif

//#define PRINT_SYNC_OR_ASYNC
#ifdef PRINT_SYNC_OR_ASYNC
#define PRINT_SYNC_ASYNC(color, type) printSyncAsync(color, type)
void printSyncAsync(const char *color, const char *type) {
  cerr << color << " " << type << RESET << endl;
}
#else
#define PRINT_SYNC_ASYNC(color, type)
#endif

//#define DEBUG_PRINTS_GPUMEMCPY
#ifdef DEBUG_PRINTS_GPUMEMCPY
#define PRINT_GPUMEMCPY(src, dst, args)                                        \
  printGPUMemcpy(src, dst, args, __func__, __FILE__, __LINE__)
void printGPUMemcpy(arax_data_s *src, arax_data_s *dst, memcpyArgs *args,
                    const char *func, const char *file, size_t line) {
  size_t sz_src = arax_data_size(src);
  size_t sz_dst = arax_data_size(dst);
  cerr << "Func: " << func << " ,file: " << file << " ,line: " << line
       << " ,src ptr: " << src << " ,src size: " << arax_data_size(src)
       << " ,dst ptr: " << dst << " ,dst size: " << arax_data_size(dst)
       << " ,src offset: " << args->src_offset
       << " ,dst offset: " << args->dst_offset
       << " ,(sz_src - src_offset): " << sz_src - args->src_offset
       << " ,(sz_dst - dst_offset): " << sz_dst - args->dst_offset
       << " , args->size: " << args->size << endl;
}
#else
#define PRINT_GPUMEMCPY(src, dst, args)
#endif

/*
 * Checks if \c data is aligned in GPU memory.
 */
void isAligned(arax_data_s *data) {
  if (((unsigned long)data->remote) % 128) {
    cerr << " Unaligned memory access: " << ((unsigned long)data->remote) % 128
         << endl;
    cerr << " ABORT!!! " << endl;
    abort();
  }
}

/*
 * Checks if a pointer is in GPU memory.
 */
void is_device_pointer(const void *ptr, int line) {
  hipPointerAttribute_t attributes;
  hipError_t err;
  err = hipPointerGetAttributes(&attributes, ptr);
  CUDA_ERROR_FATAL(err);

  if (attributes.devicePointer != NULL) {
    // cerr<<attributes.memoryType<<" Device pointer\n";
    // cerr<<" Device pointer "<<ptr<< " line: "<<line<<endl;
  } else {
    cerr << " NO Device pointer " << ptr << " line: " << line << " ABORT...\n";
    abort();
  }
}

/*
 * Checks if \c data has been allocated in GPU, if not allocates.
 * \return True if already allocated or just allocated. False if allocation
 * failed.
 */
bool alloc_no_throttle(arax_data_s *data) {
  size_t sz = arax_data_size(data);
  // data already allocated
  if (data->remote) {
    return true;
  }
  arax_assert(data->accel);
  arax_assert((arax_accel_s *)((arax_vaccel_s *)(data->accel))->phys);
  data->phys = ((arax_accel_s *)((arax_vaccel_s *)(data->accel))->phys);
  arax_accel_size_dec(((arax_vaccel_s *)(data->accel))->phys,
                      arax_data_size(data));
  // isAligned(data);

  hipError_t err = hipMalloc(&(data->remote), sz);
  PRINT_DATA_LEAKS(data);
  CUDA_ERROR_FATAL(err);

  return true;
}
//#define DEBUG_PRINTS
/*Allocates data in the GPU*/
ARAX_HANDLER_EX(alloc_data, GPU, hipStream_t *stream) {
  arax_assert(task->in_count == 0);
  arax_assert(task->out_count == 1);
  arax_data_s *data = (arax_data_s *)task->io[0];
  size_t sz = arax_data_size(data);

  // data already allocated
  if (data->remote) {
    // cerr<<"Have already remote. AllocData: "<<__FILE__<<__LINE__<<endl;
    arax_task_mark_done(task, task_completed);
    return task_completed;
  }
  arax_assert((arax_accel_s *)((arax_vaccel_s *)(data->accel))->phys);
  // isAligned(data);
  hipError_t err = hipMalloc(&(data->remote), sz);
  CUDA_ERROR_FATAL(err);
#ifdef DEBUG_PRINTS
  std::cerr << __func__ << " task: " << task << " data: " << data
            << " data remote: " << data->remote << " size: " << sz << std::endl;
#endif

  arax_assert(data->remote);
  data->phys = ((arax_accel_s *)((arax_vaccel_s *)(data->accel))->phys);

  arax_task_mark_done(task, task_completed);
  return task_completed;
}
void arax_data_memcpy_gpu_cb(void *userData) {
  arax_task_msg_s *task = (arax_task_msg_s *)userData;
  // arax_data_free(task->io[0]);
  arax_task_free(task);
}
/**
 * Performs a Remote to Remote copy
 */

ARAX_HANDLER_EX(memcpy, GPU, hipStream_t *stream) {
  memcpyArgs *args =
      (memcpyArgs *)arax_task_host_data(task, sizeof(memcpyArgs));
  arax_data_s *src = (arax_data_s *)(task->io[0]);
  arax_data_s *dst = (arax_data_s *)(task->io[1]);
#ifdef DEBUG_PRINTS
  std::cerr << __func__ << " task: " << task << " src: " << src
            << " offset: " << args->src_offset << " dst: " << dst
            << " offset: " << args->dst_offset << std::endl;
#endif

  size_t sz = args->size;
  if (args->size == 0) {
    std::cerr << __FILE__ << " " << __func__
              << " args->size is Zero. Please specify size! Abort.\n";
    abort();
  }

  hipError_t err;
  size_t sz_src = arax_data_size(src);
  size_t sz_dst = arax_data_size(dst);

  PRINT_GPUMEMCPY(src, dst, args);

  if (!alloc_no_throttle(dst)) {
    arax_task_mark_done(task, task_failed);
    return task_failed;
  }
  if (!alloc_no_throttle(src)) {
    arax_task_mark_done(task, task_failed);
    return task_failed;
  }

  arax_assert(dst->remote);
  arax_assert(src->remote);

  if ((sz_dst - (args->dst_offset) < sz)) {
    cerr << __FILE__ << " " << __LINE__ << " " << __func__ << endl;
    cerr << "arax_data_size(dst)-dst_offset: " << sz_dst - args->dst_offset
         << " is < args->size: " << sz << endl;
    abort();
  }
  if ((sz_src - (args->src_offset)) < sz) {
    cerr << __FILE__ << " " << __LINE__ << " " << __func__ << endl;
    cerr << " arax_data_size(src)-src_offset: " << sz_src - (args->src_offset)
         << " is < args->size: " << sz << endl;
    abort();
  }

  char *src_ptr = ((char *)src->remote) + args->src_offset;
  char *dst_ptr = ((char *)dst->remote) + args->dst_offset;
  // Remote to remote copy
  err =
      hipMemcpyAsync(dst_ptr, src_ptr, sz, hipMemcpyDeviceToDevice, *stream);

#ifdef DEBUG
  float *src_h = 0, *dst_h = 0;
  src_h = (float *)malloc(sz);
  dst_h = (float *)malloc(sz);

  hipMemcpy(src_h, ((char *)src->remote) + args->src_offset, sz,
             hipMemcpyDeviceToHost);
  hipMemcpy(dst_h, ((char *)dst->remote) + args->dst_offset, sz,
             hipMemcpyDeviceToHost);

  std::cerr << __func__ << " AFTER task: " << task << " dst: " << dst
            << " dst value: " << src_h[0] << " src: " << src
            << " src value: " << dst_h[0] << std::endl;
#endif

  CUDA_ERROR_FATAL(err);

  // If task is async free task from controller
  if (args->sync == false) {
    err = hipStreamSynchronize(*stream);
    CUDA_ERROR_FATAL(err);
    return task_completed;
  }
  arax_task_mark_done(task, task_completed);
  return task_completed;
}

/**
 * Performs a Memset
 */
void arax_data_memset_gpu_cb(void *userData) {
  arax_task_msg_s *task = (arax_task_msg_s *)userData;
  arax_task_mark_done(task, task_completed);
  arax_task_free(task);
}
ARAX_HANDLER_EX(memset, GPU, hipStream_t *stream) {
  hipError_t err;
  memsetArgs *args =
      (memsetArgs *)arax_task_host_data(task, sizeof(memsetArgs));
  arax_data_s *data = (arax_data_s *)(task->io[0]);

  if (!alloc_no_throttle(data)) {
    arax_task_mark_done(task, task_failed);
    return task_failed;
  }
  arax_assert(data->remote);
  char *dst = ((char *)data->remote) + args->data_offset;
  err = hipMemsetAsync((void *)dst, args->value, args->size, *stream);
#ifdef DEBUG_PRINTS
  std::cerr << __func__ << " task: " << task << " data: " << data
            << " remote: " << data->remote << " offset: " << args->data_offset
            << std::endl;
#endif

  CUDA_ERROR_FATAL(err);
  arax_task_mark_done(task, task_completed);
  return task_completed;
}

ARAX_HANDLER_EX(free, GPU, hipStream_t *stream) {
  void **args = (void **)arax_task_host_data(task, sizeof(void *) * 4);
  void *ptrAtDevice = args[1];
  size_t size = (size_t)args[2];
  arax_vaccel_s *accel_ptr = (arax_vaccel_s *)args[3];
#ifdef DEBUG_PRINTS
  void *host = args[0];
  std::cerr << __func__ << " task: " << task << " data: " << host
            << " remote: " << ptrAtDevice << " VAQ phys: " << accel_ptr
            << std::endl;
#endif
  hipError_t err = hipFree(ptrAtDevice);
  arax_assert(accel_ptr);
  // increment data->phys
  arax_accel_size_inc(accel_ptr, size);

  arax_task_mark_done(task,
                      (err != hipSuccess) ? task_failed : task_completed);

  CUDA_ERROR_FATAL(err);

  arax_task_free(task);

  return task_completed;
}

ARAX_HANDLER_EX(init_phys, GPU, hipStream_t *stream) {
  hipStreamSynchronize(*stream);
  arax_task_mark_done(task, task_completed);
  return task_completed;
}

void arax_data_set_gpu_cb(void *userData) {
  arax_task_msg_s *task = (arax_task_msg_s *)userData;
  arax_task_mark_done(task, task_completed);
  arax_task_free(task);
}

ARAX_HANDLER_EX(arax_data_set, GPU, hipStream_t *stream) {
  hipError_t err;
  arax_assert(task->in_count == 0);
  arax_assert(task->out_count == 1);
  void *host_src = arax_task_host_data(task, task->host_size);
  arax_data_s *data = (arax_data_s *)(task->io[0]);
  if (!alloc_no_throttle(data)) {
    arax_task_mark_done(task, task_failed);
    return task_failed;
  }
#ifdef DEBUG_PRINTS
  std::cerr << "H2D Size: " << arax_data_size(data) << std::endl;
  std::cerr << __func__ << " task: " << task << " data: " << data
            << " remote: " << data->remote << " data: " << data
            << " size: " << task->host_size << " host: " << host_src
            << std::endl;
#endif
//#define BREAKDOWNS
#ifdef BREAKDOWNS
  auto start_1 = std::chrono::high_resolution_clock::now();
#endif
  arax_assert(data->remote);
  err = hipMemcpyAsync(data->remote, host_src, task->host_size,
                        hipMemcpyDefault, *stream);
  CUDA_ERROR_FATAL(err);
  hipStreamSynchronize(*stream);
#ifdef BREAKDOWNS
  auto end_1 = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> elapsed_milli1 = end_1 - start_1;
  std::cerr << "H2D time : " << elapsed_milli1.count() << " ms" << std::endl;
#endif
  arax_task_free(task);
  return task_completed;
}

void arax_data_get_gpu_cb(void *userData) {
  arax_task_msg_s *task = (arax_task_msg_s *)userData;
  arax_task_mark_done(task, task_completed);
}

ARAX_HANDLER_EX(arax_data_get, GPU, hipStream_t *stream) {
  hipError_t err;
  arax_assert(task->in_count == 0);
  arax_assert(task->out_count == 1);
  void *host_src = arax_task_host_data(task, task->host_size);
  arax_data_s *data = (arax_data_s *)(task->io[0]);
  arax_assert(data->remote);
  err = hipMemcpyAsync(host_src, data->remote, arax_data_size(data),
                        hipMemcpyDefault, *stream);

  CUDA_ERROR_FATAL(err);
#ifdef DEBUG_PRINTS
  std::cerr << "D2H Size: " << arax_data_size(data) << std::endl;
  std::cerr << __func__ << " task: " << task << " data: " << data
            << " remote: " << data->remote << std::endl;
#endif
  hipStreamSynchronize(*stream);
  arax_task_mark_done(task, task_completed);
  return task_completed;
}
